#include "hip/hip_runtime.h"

#include <assert.h>
#include <iostream>
#include <memory>
#include <stdio.h>
#include <hip/hip_runtime.h>

#include "cuda/api_wrappers.hpp"  // -Icuda-api-wrappers/src/

// ================================================================ utils

// #define GPU_ERR_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }

// inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
// {
//    if (code != hipSuccess)
//    {
//       fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
//       if (abort) exit(code);
//    }
// }


// #define CUDA_MALLOC()



// ================================================================ kernels

// template<class DataT> __global__ void
// vectorAdd(const DataT* A, const DataT* B, DataT* C, int numElements)
__global__ void
vectorAdd(const float* A, const float* B, float* C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

// template void vectorAdd<float>(const float* A, const float* B, float* C, int numElements);

// ================================================================ main

int main(int argc, const char* argv[]) {

    // hipError_t err = hipSuccess;
    using DataT = float;
    static const int nelements = 50*1000;
    static const int nbytes = nelements * sizeof(DataT);

    // DataT* a_h = (DataT*)malloc(nbytes);
    // DataT* b_h = (DataT*)malloc(nbytes);
    // DataT* c_h = (DataT*)malloc(nbytes);
    auto a_h = std::make_unique<DataT[]>(nbytes);
    auto b_h = std::make_unique<DataT[]>(nbytes);
    auto c_h = std::make_unique<DataT[]>(nbytes);

    // initialize inputs
    for (int i = 0; i < nelements; i++) {
        a_h[i] = 2 * i;
        b_h[i] = nelements - i;
    }

    // alloc device vecs

    // DataT* a_d = nullptr;
    // DataT* b_d = nullptr;
    // DataT* c_d = nullptr;
    // err = hipMalloc((void**)&a_d, sz);
    // assert(err == hipSuccess);
    // err = hipMalloc((void**)&b_d, sz);
    // assert(err == hipSuccess);

    // using namespace cuda::device; // does this compile?
    // using namespace cuda::device::current; // does this compile?

    auto dev = cuda::device::current::get();
    auto a_d = cuda::memory::device::make_unique<DataT[]>(dev, nelements);
    auto b_d = cuda::memory::device::make_unique<DataT[]>(dev, nelements);
    auto c_d = cuda::memory::device::make_unique<DataT[]>(dev, nelements);

    cuda::memory::copy(a_d.get(), a_h.get(), nbytes);
    cuda::memory::copy(b_d.get(), b_h.get(), nbytes);

    int threadsPerBlock = 256;
    int blocksPerGrid = (nelements + threadsPerBlock - 1) / threadsPerBlock;
    std::cout << "CUDA kernel launch with " << blocksPerGrid
        << " blocks of " << threadsPerBlock << " threads\n";

    cuda::launch(vectorAdd,
        {blocksPerGrid, threadsPerBlock},
        a_d.get(), b_d.get(), c_d.get(), nelements);

    cuda::memory::copy(c_h.get(), c_d.get(), nbytes);

    printf("vec_add: main done\n");
    return 0;
}
